#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "superimpose.h"

__global__ void overlayKernel(unsigned char* largeImage, unsigned char* smallImage, int largeWidth, int largeHeight, int smallWidth, int smallHeight, int startX, int startY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= startX && x < (startX + smallWidth) && y >= startY && y < (startY + smallHeight)) {
        int largeOffset = (y * largeWidth + x) * 4;
        int smallOffset = ((y - startY) * smallWidth + (x - startX)) * 4;
        
        for (int i = 0; i < 4; ++i) {
            largeImage[largeOffset + i] = smallImage[smallOffset + i];
        }
    }
}

void overlayImages(unsigned char* largeImage, unsigned char* smallImage, int largeWidth, int largeHeight, int smallWidth, int smallHeight, int startX, int startY) {
    unsigned char *d_largeImage, *d_smallImage;
    hipMalloc((void**)&d_largeImage, largeWidth * largeHeight * 4);
    hipMalloc((void**)&d_smallImage, smallWidth * smallHeight * 4);

    hipMemcpy(d_largeImage, largeImage, largeWidth * largeHeight * 4, hipMemcpyHostToDevice);
    hipMemcpy(d_smallImage, smallImage, smallWidth * smallHeight * 4, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((largeWidth + blockDim.x - 1) / blockDim.x, (largeHeight + blockDim.y - 1) / blockDim.y);

    overlayKernel<<<gridDim, blockDim>>>(d_largeImage, d_smallImage, largeWidth, largeHeight, smallWidth, smallHeight, startX, startY);
    hipDeviceSynchronize();

    hipMemcpy(largeImage, d_largeImage, largeWidth * largeHeight * 4, hipMemcpyDeviceToHost);

    hipFree(d_largeImage);
    hipFree(d_smallImage);
}
